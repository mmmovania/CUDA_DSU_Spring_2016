#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include <stdio.h>
#include <stdlib.h>
#include "common/book.h"
#include "common/cpu_bitmap.h"

const int WIDTH = 512;
const int HEIGHT = 512;

__global__ void ConvolutionKernel(unsigned char* d_Pin,
                             unsigned char* d_Pout,
                             int width, int height) { 
   //Calculate the row # 
   int y = blockIdx.y*blockDim.y + threadIdx.y; 

   //Calculate the column # 
   int x = blockIdx.x*blockDim.x + threadIdx.x; 
   
   int index = x + (height-1-y)*width;

   float kernel[3][3]={{-1,-1,-1},
		       {-1,9,-1},
		       {-1,-1,-1}};

   float sum = 0;
   
   for(int k=-1;k<=1;++k) {
      for(int m=-1;m<=1;++m) {
         
         int f = x + m;
         int g = y + k;
         if(f>0 && f < width && g>0 && g < height) { 
            int index1 = f+g*width;
            sum = sum + kernel[k+1][m+1]*d_Pin[index1];
         }
      }
   }

   d_Pout[index] = sum;
} 

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

void loadBinImage(const char* imageName, unsigned char* pData) {

   FILE* fp = fopen(imageName, "rb");
   fread(pData, 1, WIDTH*HEIGHT, fp);
   fclose(fp);
}

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap(WIDTH, HEIGHT, &data );
    unsigned char    *host_bitmap;
    unsigned char    *dev_bitmap;
    unsigned char    *dev_bitmap2;

    host_bitmap = (unsigned char*)malloc(bitmap.image_size() );
    memset(host_bitmap,0, bitmap.image_size());
   
    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap2, bitmap.image_size() ) );

    HANDLE_ERROR( hipMemcpy( dev_bitmap2, host_bitmap, 
                              bitmap.image_size(),
                              hipMemcpyHostToDevice ) ); 

    loadBinImage("Baboon.raw", host_bitmap);
    HANDLE_ERROR( hipMemcpy( dev_bitmap, host_bitmap, 
                              bitmap.image_size(),
                              hipMemcpyHostToDevice ) ); 
   
    data.dev_bitmap = dev_bitmap;

    dim3    blocksGrid;
    dim3    threadsBlock(16,16,1);
    blocksGrid.x = ceil(WIDTH/16.0);
    blocksGrid.y = ceil(HEIGHT/16.0);
 
    ConvolutionKernel<<<blocksGrid, threadsBlock>>>( dev_bitmap, dev_bitmap2, WIDTH, HEIGHT);

    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap2,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );
                              
    HANDLE_ERROR( hipFree( dev_bitmap ) );
    HANDLE_ERROR( hipFree( dev_bitmap2 ) );
    free(host_bitmap);                              
    bitmap.display_and_exit();
}

