
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Add(int* a, int* b, int* c) {
   *c = *a + *b;
}

int main()
{
   //host memory
   int h_a = 10;
   int h_b = 20;
   int h_c =  0;

   //device memory pointers
   int* d_a = 0;
   int* d_b = 0;
   int* d_c = 0;

   //allocate device memory
   hipMalloc(&d_a, sizeof(int));
   hipMalloc(&d_b, sizeof(int));
   hipMalloc(&d_c, sizeof(int));

   //copy data from host to device memory
   hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

   //launch kernel
   Add<<<1,1>>>(d_a, d_b, d_c);
   hipDeviceSynchronize();

   //copy data from device to host memory
   hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

   //output result
   printf("Result: %3d \n", h_c);

   //release device memory
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   hipDeviceReset();

   return 0;
}